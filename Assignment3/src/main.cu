#include "hip/hip_runtime.h"
#include <time.h>
#include <glew.h>
#include <freeglut.h>
#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>
#include <string>
#include <random>
#include <math.h>
#include "ball.h"

#define WWIDTH 800
#define WHEIGHT 800
#define UPDATE_FREQ 1.f / 30
#define BALL_COUNT 10
#define MIN_RADIUS 0.05f
#define PI 3.141592f
#define DEGREE_TO_RAD PI / 180
#define NUM_POINTS 360
#define BLOCK_SIZE 256

ball* balls, *d_balls;
ball** d_pairs;
size_t balls_count, pairs_count;
size_t balls_size, pairs_size;
size_t balls_grid_size, pairs_grid_size = 0;

vector2d GRAVITY(0.f, -1.5f);

clock_t previous_t = 0, current_t = 0;
float delta_t = UPDATE_FREQ;

__global__ void init_pairs(ball** d_pairs, ball* d_balls, size_t balls_count) {
	int count = 0;
	for (unsigned int i = 0; i < balls_count; ++i) {
		for (unsigned int j = (i + 1); j < balls_count; ++j) {
			d_pairs[count++] = &d_balls[i];
			d_pairs[count++] = &d_balls[j];
		}
	}
}

void init(int argc, char **argv) {
	//////////////////////////init display//////////////////////////
	glutInit(&argc, argv);
	glutInitWindowPosition(-1, -1);
	glutInitWindowSize(WWIDTH, WHEIGHT);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_ALPHA);
	glutCreateWindow("Bouncing Balls Simulation");
	glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glEnable(GL_BLEND);
	////////////////////////////////////////////////////////////////

	///////////////////////////init balls///////////////////////////
	balls_count = argc > 1 ? std::stoi(argv[1]) : BALL_COUNT;
	balls = new ball[balls_count];

	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<int> rad(1, 3);
	std::uniform_real_distribution<float> vel(-1.f, 1.f);

	for (unsigned int i = 0; i < balls_count; ++i) {
		int test = rad(gen);
		float radius = MIN_RADIUS * test; // random radius

		float ur_bound = radius - 1;
		float ll_bound = 1 - radius;

		std::uniform_real_distribution<float> coord(ur_bound, ll_bound); // so we dont get balls out of bounds
		vector2d center(coord(gen), coord(gen));

		int weight = (int)(radius * 100.0f);
		vector2d velocity(vel(gen), vel(gen));
		balls[i] = ball(radius, center, velocity, weight);
	}

	balls_size = balls_count * sizeof(ball);
	hipMalloc(&d_balls, balls_size);
	hipMemcpy(d_balls, balls, balls_size, hipMemcpyHostToDevice);
	balls_grid_size = (size_t)std::ceilf((float)balls_count / BLOCK_SIZE);
	////////////////////////////////////////////////////////////////

	///////////////////////////init pairs///////////////////////////
	for (unsigned int i = 0; i < balls_count; ++i) {
		for (unsigned int j = (i + 1); j < balls_count; ++j) {
			++pairs_count;
		}
	}

	hipMalloc(&d_pairs, pairs_count * sizeof(ball*) * 2);
	init_pairs<<<1, 1>>>(d_pairs, d_balls, balls_count);
	pairs_grid_size = (size_t)std::ceilf((float)pairs_count / BLOCK_SIZE);
	////////////////////////////////////////////////////////////////
}

__global__ void wall_bounce(ball* d_balls, float delta_t, size_t balls_count, vector2d GRAVITY) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < balls_count) {
		ball& current = d_balls[id];

		current.velocity.y += delta_t * GRAVITY.y;
		current.center.x += delta_t * current.velocity.x;
		current.center.y += delta_t * current.velocity.y;

		float t_wall = 1.f - current.radius;
		float b_wall = current.radius - 1.f;
		float r_wall = t_wall;
		float l_wall = b_wall;

		if (current.center.x > r_wall) {
			current.center.x = r_wall;
			current.velocity.x *= -1.f;
		}
		else if (current.center.x < l_wall) {
			current.center.x = l_wall;
			current.velocity.x *= -1.f;
		}

		if (current.center.y > t_wall) {
			current.center.y = t_wall;
			current.velocity.y *= -1.f;
		}
		else if (current.center.y < b_wall) {
			current.center.y = b_wall;
			current.velocity.y *= -1.f;
		}
	}
}

__global__ void ball_bounce(ball** d_pairs, size_t pairs_count) {
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < pairs_count) {
		unsigned int stride = 2 * id;
		ball* current = d_pairs[stride];
		ball* other = d_pairs[stride + 1];

		float min_dist = current->radius + other->radius;

		// check for aabb overlap
		// if true, balls are close enough, computation is worth it.
		if (current->center.x + min_dist > other->center.x
			&& current->center.y + min_dist > other->center.y
			&& other->center.x + min_dist > current->center.x
			&& other->center.y + min_dist > current->center.y) {

			float c_x = current->center.x - other->center.x;
			float c_y = current->center.y - other->center.y;
			float c = powf(c_x, 2.f) + powf(c_y, 2.f);

			// balls are close enough, but it does not mean they have collided.
			// check for ball collision.
			// if true, collision occured, handle it
			if (c <= powf(min_dist, 2.f)) {
				float distance = sqrtf(c);
				float overlap = 0.5f * (distance - current->radius - other->radius);

				float dir_x = c_x / distance;
				float dir_y = c_y / distance;

				current->center.x -= overlap * dir_x;
				current->center.y -= overlap * dir_y;
				other->center.x += overlap * dir_x;
				other->center.y += overlap * dir_y;

				float v_x = current->velocity.x - other->velocity.x;
				float v_y = current->velocity.y - other->velocity.y;
				int m = current->mass + other->mass;
				float mag = powf(distance, 2.f);
				float dot_vc = v_x * c_x + v_y * c_y;
				float ratio = 2.f * dot_vc / (m * mag);

				current->velocity.x -= (other->mass * ratio * c_x);
				current->velocity.y -= (other->mass * ratio * c_y);
				other->velocity.x += (current->mass * ratio * c_x);
				other->velocity.y += (current->mass * ratio * c_y);
			}
		}
	}
}

void draw() {
	glClearColor(0.25f, 0.25f, 0.25f, 1.f);
	glClear(GL_COLOR_BUFFER_BIT);

	for (int i = 0; i < balls_count; ++i) {
		ball& ball = balls[i];

		glBegin(GL_POLYGON);
		glColor4f(ball.color.x, ball.color.y, ball.color.z, 0.25f);

		for (int i = 0; i < NUM_POINTS; ++i) {
			float angle = i * DEGREE_TO_RAD;

			glVertex2d
			(
				ball.radius * cos(angle) + ball.center.x,	// x-coord
				ball.radius * sin(angle) + ball.center.y	// y-coord
			);
		}

		glEnd();
	}
	glutSwapBuffers();
}

void update() {
	//update current clock time
	current_t = clock();
	delta_t = (float)(current_t - previous_t) / CLOCKS_PER_SEC;

	// don't draw if delta_t is faster than 30 fps
	if (delta_t < UPDATE_FREQ) return;

	// store last draw time
	previous_t = current_t;

	wall_bounce<<<balls_grid_size, BLOCK_SIZE>>>(d_balls, delta_t, balls_count, GRAVITY);
	ball_bounce<<<pairs_grid_size, BLOCK_SIZE >>>(d_pairs, pairs_count);
	hipMemcpy(balls, d_balls, balls_size, hipMemcpyDeviceToHost);

	draw();
}

int main(int argc, char **argv) {
	init(argc, argv);
	glutDisplayFunc(update);
	glutIdleFunc(update);
	glutMainLoop();

	delete[] balls;
	hipFree(d_balls);
	hipFree(d_pairs);

	return 0;
}